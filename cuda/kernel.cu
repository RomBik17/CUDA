#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

#define N 10

void printDevProp()
{
    hipDeviceProp_t devProp;
    int count;
    hipGetDeviceCount(&count);
    hipGetDeviceProperties(&devProp, count - 1);

    printf("Major revision number:         %d\n", devProp.major);
    printf("Minor revision number:         %d\n", devProp.minor);
    printf("Name:                          %s\n", devProp.name);
    printf("Total global memory:           %u\n", devProp.totalGlobalMem);
    printf("Total shared memory per block: %u\n", devProp.sharedMemPerBlock);
    printf("Total registers per block:     %d\n", devProp.regsPerBlock);
    printf("Warp size:                     %d\n", devProp.warpSize);
    printf("Maximum memory pitch:          %u\n", devProp.memPitch);
    printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
    for (int i = 0; i < 3; ++i)
        printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
    printf("Clock rate:                    %d\n", devProp.clockRate);
    printf("Total constant memory:         %u\n", devProp.totalConstMem);
    printf("Texture alignment:             %u\n", devProp.textureAlignment);
    printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
    printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
    printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));

    return;
}

//vector add function
__global__ void addKernel(int *a, int *b, int* c)
{
    int tID = blockIdx.x;
    if (tID < N)
        c[tID] = a[tID] + b[tID];
}

int main()
{
    printDevProp();
    int a[N], b[N], c[N];
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));

    for (int i = 0; i < N; ++i)
    {
        a[i] = -i;
        b[i] = i * i;
    }

    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    addKernel << <N, 1 >> > (dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    
    for (int i = 0; i < N; ++i)
    {
        printf("%d\n", c[i]);
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
    return 0;
}
